#include "hip/hip_runtime.h"
#define BLOCK_SIZE 64

#define _DEBUG
#include <stdio.h>
#include "caffe/util/modified_permutohedral.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/util/hash_helper.cu"

namespace caffe {

template<int pd>
__global__ static void createMatrix(int num_points,
				    const float *positions,
				    int *table_entries,
				    int table_capacity,
				    signed short* table_keys,				    
				    const float *scaleFactor,
				    MatrixEntry *matrix)
{
    // scanline order
    const int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    const bool outOfBounds = (idx>=num_points) ;
    // TODO : change that!!!
    const int threadId = idx;

    // 8x8 blocks
    //const int x = threadIdx.x + blockIdx.x * blockDim.x;
    //const int y = threadIdx.y + blockIdx.y * blockDim.y;
    //const int threadId = threadIdx.y*blockDim.x + threadIdx.x;
    //const int idx = y*w + x;
    //const bool outOfBounds = (x >= w) || (y >= h);
  
    float myElevated[pd+1];
    const float *myPosition = positions + idx*pd;

    int myGreedy[pd+1];
    int myRank[pd+1];

    float myBarycentric[pd+2];
    __shared__ short keys[pd*BLOCK_SIZE];
    short *myKey = keys + threadId * pd;

    if (!outOfBounds) {

	myElevated[pd] = -pd*(myPosition[pd-1])*scaleFactor[pd-1];
	for (int i = pd-1; i > 0; i--) {
	    myElevated[i] = (myElevated[i+1] -
			     i*(myPosition[i-1])*scaleFactor[i-1] +
			     (i+2)*(myPosition[i])*scaleFactor[i]);
	}
	myElevated[0] = myElevated[1] + 2*(myPosition[0])*scaleFactor[0];


	// find the closest zero-colored lattice point

	// greedily search for the closest zero-colored lattice point
	signed short sum = 0;
	for (int i = 0; i <= pd; i++) {
	    float v = myElevated[i]*(1.0f/(pd+1));
	    float up = ceilf(v) * (pd+1);
	    float down = floorf(v) * (pd+1);
	    if (up - myElevated[i] < myElevated[i] - down) {
		myGreedy[i] = (signed short)up;
	    } else {
		myGreedy[i] = (signed short)down;
	    }
	    sum += myGreedy[i];
	}
	sum /= pd+1;

	// sort differential to find the permutation between this simplex and the canonical one
	for (int i = 0; i <= pd; i++) {
	    myRank[i] = 0;
	    for (int j = 0; j <= pd; j++) {
		if (myElevated[i] - myGreedy[i] < myElevated[j] - myGreedy[j] ||
		    (myElevated[i] - myGreedy[i] == myElevated[j] - myGreedy[j]
		     && i > j)) {
		    myRank[i]++;
		}
	    }
	}

	if (sum > 0) { // sum too large, need to bring down the ones with the smallest differential
	    for (int i = 0; i <= pd; i++) {
		if (myRank[i] >= pd + 1 - sum) {
		    myGreedy[i] -= pd+1;
		    myRank[i] += sum - (pd+1);
		} else {
		    myRank[i] += sum;
		}
	    }
	} else if (sum < 0) { // sum too small, need to bring up the ones with largest differential
	    for (int i = 0; i <= pd; i++) {
		if (myRank[i] < -sum) {
		    myGreedy[i] += pd+1;
		    myRank[i] += (pd+1) + sum;
		} else {
		    myRank[i] += sum;
		}
	    }
	}

	// turn delta into barycentric coords
	for (int i = 0; i <= pd+1; i++) {
	    myBarycentric[i] = 0;
	}

	for (int i = 0; i <= pd; i++) {
	    float delta = (myElevated[i] - myGreedy[i]) * (1.0f/(pd+1));
	    myBarycentric[pd-myRank[i]] += delta;
	    myBarycentric[pd+1-myRank[i]] -= delta;
	}
	myBarycentric[0] += 1.0f + myBarycentric[pd+1];
    }

    #ifdef USE_ADDITIVE_HASH
    unsigned int cumulative_hash = hash(myGreedy, pd);
    #endif
    for (int color = 0; color <= pd; color++) {
	// Compute the location of the lattice point explicitly (all but
	// the last coordinate - it's redundant because they sum to zero)
	if (!outOfBounds) {
	    for (int i = 0; i < pd; i++) {
		myKey[i] = myGreedy[i] + color;
		if (myRank[i] > pd-color) myKey[i] -= (pd+1);
	    }
	}

	#ifdef USE_ADDITIVE_HASH
	for (int i = 0; i < pd; i++) {
	    if (myRank[i] == pd-color) cumulative_hash += hOffset[i];
	}
	#endif

	if (!outOfBounds) {
	    MatrixEntry r;
	    #ifdef USE_ADDITIVE_HASH
	    r.index = hashTableInsert(cumulative_hash, myKey, table_keys,
    		table_entries, table_capacity,  idx*(pd+1)+color,pd);
	    #else
	    r.index = hashTableInsert(myKey, table_keys, table_entries,
    		table_capacity,  idx*(pd+1)+color,pd);
	    #endif
	    r.weight = myBarycentric[color];
	    matrix[idx*(pd+1) + color] = r;
	}
    }
}

template<int kd>
__global__ static void cleanHashTable(int n,
				    int *table_entries,
				    int table_capacity,
				    signed short* table_keys,
				    MatrixEntry *matrix)
{
    const int idx = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.x;

    if (idx >= n) return;

    // find my hash table entry
    int *e = table_entries + idx;

    // Check if I created my own key in the previous phase
    if (*e >= 0) {
	// Rehash my key and reset the pointer in order to merge with
	// any other pixel that created a different entry under the
	// same key. If the computation was serial this would never
	// happen, but sometimes race conditions can make the same key
	// be inserted twice. hashTableRetrieve always returns the
	// earlier, so it's no problem as long as we rehash now.
	*e = hashTableRetrieve(table_keys + *e*kd,
	        table_entries, table_keys, table_capacity, kd);
    }
}

template<int pd>
void gpu_init(const float* features, HashTable table, MatrixEntry* matrix, int num_points)
{
    unsigned int blocks = (num_points-1)/64 + 1;
    unsigned int blockSize = 64;
    float blurVariance = 0.5 ;
    float * scaleFactor;
    float* scaleFactorHost = new float[pd];
    
    // Create Scale factor vector and give it to GPU
    // num_dimensions is likely to be low so do that 
    // on the CPU
    for (int i = 0; i < pd; i++) {
	scaleFactorHost[i] = (pd+1)*sqrtf((1.0/6 + blurVariance)/((i+1)*(i+2)));
    }
    CUDA_CHECK(hipMalloc((void**)&scaleFactor, sizeof(float)*pd));
    CUDA_CHECK(hipMemcpy(scaleFactor, scaleFactorHost, sizeof(float)*pd, hipMemcpyHostToDevice));
    
    // Allocate matrix
    CUDA_CHECK(hipMalloc((void **)&matrix, sizeof(MatrixEntry)*(num_points*(pd+1))));
    

    // Populate memory for hash helpers
    unsigned long long int __host_two32 = ((unsigned long long int)1)<<32;
    unsigned int __host_div_c = 2*(num_points*(pd+1));
    unsigned int __host_div_l = ceilf(logf((float)__host_div_c) / logf(2.0f));
    unsigned int __host_div_m = (__host_two32<<__host_div_l)/__host_div_c - __host_two32 + 1;
    /*CUDA_CHECK(hipMemcpy((char*)&__div_c, &__host_div_c, sizeof(unsigned int)));
    CUDA_CHECK(hipMemcpy((char*)&__div_l, &__host_div_l, sizeof(unsigned int)));
    CUDA_CHECK(hipMemcpy((char*)&__div_m, &__host_div_m, sizeof(unsigned int)));

    // Populate constant memory with hash of offset vectors
    unsigned int hOffset_host[num_dimensions+1];
    signed short offset[num_dimensions+1];
    for (int i = 0; i < num_dimensions; offset[i] = 1, i++);
    for (int i = 0; i <= num_dimensions; i++) {
      offset[i] -= num_dimensions+1; hOffset_host[i] = hash<num_dimensions>(offset); offset[i] += num_dimensions+1;
    }
    CUDA_CHECK(hipMemcpyToSymbol((char*)&hOffset, &hOffset_host, sizeof(unsigned int)*(num_dimensions+1)));
*/

    createMatrix<pd><<<blocks, blockSize>>>(num_points,
    					    features,
    					    table.table_entries,
    					    table.table_capacity,
    					    table.table_keys,
					    scaleFactor,
					    matrix);
    CUDA_POST_KERNEL_CHECK;

    // fix duplicate hash table entries
    int cleanBlockSize = 32;
    dim3 cleanBlocks((num_points-1)/cleanBlockSize+1, 2*(pd+1), 1);
    cleanHashTable<pd><<<cleanBlocks, cleanBlockSize>>>(2*num_points*(pd+1),
         table.table_entries, table.table_capacity, table.table_keys,
         matrix);
    CUDA_POST_KERNEL_CHECK;
    
    // Clean intermediate variables
    // TODO : see what can be further cleaned
    delete[] scaleFactorHost;
    CUDA_CHECK(hipFree(scaleFactor));
}

void ModifiedPermutohedral::init_gpu(const float* features, int num_dimensions, int num_points) {
  //Initialize Hash table
  table.createHashTable(num_points*(num_dimensions+1), num_dimensions, 1);
  switch(num_dimensions){
    case 2:
      gpu_init<2>(features, table, matrix,  num_points);
      break;
    case 5:
      gpu_init<5>(features, table, matrix, num_points);
    default:
      LOG(FATAL) << "num_dimensions should be 2 or 5";
  } 
}

}//namespace caffe
