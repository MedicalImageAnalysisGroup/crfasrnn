#include "hip/hip_runtime.h"
/*!
 *  \brief     The Caffe layer that implements the CRF-RNN described in the paper:
 *             Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *  \authors   Sadeep Jayasumana, Bernardino Romera-Paredes, Shuai Zheng, Zhizhong Su.
 *  \version   1.0
 *  \date      2015
 *  \copyright Torr Vision Group, University of Oxford.
 *  \details   If you use this code, please consider citing the paper:
 *             Shuai Zheng, Sadeep Jayasumana, Bernardino Romera-Paredes, Vibhav Vineet, Zhizhong Su, Dalong Du,
 *             Chang Huang, Philip H. S. Torr. Conditional Random Fields as Recurrent Neural Networks. IEEE ICCV 2015.
 *
 *             For more information about CRF-RNN, please visit the project website http://crfasrnn.torr.vision.
 */
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {
  
template <typename Dtype>
__global__ void  computeBilateralKernel(const  int num_pixels_, 
    const Dtype* const rgb_blob, 
    const int width_, const int height_, const int channels_,
    float theta_alpha_, float theta_beta_,
    const int n, float* const output_kernel) {
  int offset = ((n * channels_ ) * height_) * width_ ;
  CUDA_KERNEL_LOOP(p, num_pixels_) {
    output_kernel[5 * p] = (float)(p % width_) / theta_alpha_;
    output_kernel[5 * p + 1] = (float)(p / width_) / theta_alpha_;
    const Dtype * const rgb_data_start = rgb_blob + offset;
    output_kernel[5 * p + 2] = (float)(rgb_data_start[p] / theta_beta_);
    output_kernel[5 * p + 3] = (float)((rgb_data_start + num_pixels_)[p] / theta_beta_);
    output_kernel[5 * p + 4] = (float)((rgb_data_start + num_pixels_ * 2)[p] / theta_beta_);
  }
}

template <typename Dtype>
__global__ void  computeSpatialKernel(const int num_pixels_,
    float* const output_kernel,
    float theta_gamma_, int width_) {
  CUDA_KERNEL_LOOP(p, num_pixels_) {
    output_kernel[2*p] = (float)(p % width_) / theta_gamma_;
    output_kernel[2*p + 1] = (float)(p / width_) / theta_gamma_;
  }
}

template <typename Dtype>
void MultiStageMeanfieldLayer<Dtype>::Reshape(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
	// Do nothing.
}


/**
 * Performs filter-based mean field inference given the image and unaries.
 *
 * bottom[0] - Unary terms
 * bottom[1] - Softmax input/Output from the previous iteration (a copy of the unary terms if this is the first stage).
 * bottom[2] - RGB images
 *
 * top[0] - Output of the mean field inference (not normalized).
 */
template <typename Dtype>
void MultiStageMeanfieldLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  const Dtype* bottom_data = bottom->gpu_data() ;
  // TODO is it suppose to be constant ?
  split_layer_bottom_vec_[0] = bottom[0]->mutable_gpu_data();
  split_layer_->Forward(split_layer_bottom_vec_, split_layer_top_vec_);

  // Initialize the bilateral lattices.
  // TODO : here ?
  bilateral_lattices_.resize(num_);
  for (int n = 0; n < num_; ++n) {
    // TODO get method for permuthohedral
    computeBilateralKernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        num_pixels_, bottom_data, width_, height_, channels_,
        theta_alpha_, theta_beta_, n,
         bilateral_kernel_buffer_.get());
    //TODO reset on GPUs ? init on GPU as well
    bilateral_lattices_[n].reset(new ModifiedPermutohedral());
    bilateral_lattices_[n]->init(bilateral_kernel_buffer_.get(), 5, num_pixels_);

    // Calculate bilateral filter normalization factors.
    // is it efficient ? yes
    Dtype* norm_output_data = bilateral_norms_.mutable_gpu_data() + bilateral_norms_.offset(n);
    // TODO compute 
    bilateral_lattices_[n]->compute(norm_output_data, norm_feed_.get(), 1);
    // TODO : do that on the GPU
    for (int i = 0; i < num_pixels_; ++i) {
      norm_output_data[i] = 1.f / (norm_output_data[i] + 1e-20f);
    }
  }

  for (int i = 0; i < num_iterations_; ++i) {
    //TODO : GPU
    meanfield_iterations_[i]->PrePass(this->blobs_, &bilateral_lattices_, &bilateral_norms_);

    meanfield_iterations_[i]->Forward_gpu();
  }
}

/**
 * Backprop through filter-based mean field inference.
 */
template<typename Dtype>
void MultiStageMeanfieldLayer<Dtype>::Backward_cpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {

  for (int i = (num_iterations_ - 1); i >= 0; --i) {
    meanfield_iterations_[i]->Backward_cpu();
  }

  vector<bool> split_layer_propagate_down(1, true);
  split_layer_->Backward(split_layer_top_vec_, split_layer_propagate_down, split_layer_bottom_vec_);

  // Accumulate diffs from mean field iterations.
  for (int blob_id = 0; blob_id < this->blobs_.size(); ++blob_id) {

    Blob<Dtype>* cur_blob = this->blobs_[blob_id].get();

    if (this->param_propagate_down_[blob_id]) {

      caffe_set(cur_blob->count(), Dtype(0), cur_blob->mutable_cpu_diff());

      for (int i = 0; i < num_iterations_; ++i) {
        const Dtype* diffs_to_add = meanfield_iterations_[i]->blobs()[blob_id]->cpu_diff();
        caffe_axpy(cur_blob->count(), Dtype(1.), diffs_to_add, cur_blob->mutable_cpu_diff());
      }
    }
  }
}



INSTANTIATE_LAYER_GPU_FUNCS(MultiStageMeanfieldLayer);

}  // namespace caffe
